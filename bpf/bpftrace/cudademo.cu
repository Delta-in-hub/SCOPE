//#! nvcc -cudart=shared -o cudademo cudademo.cu && ./cudademo

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>


// CUDA核函数，用于执行数组相加
__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

// 检查CUDA运行时API调用是否成功
inline void checkCudaError(hipError_t err, const char* msg)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA错误: " << msg << " (" << hipGetErrorString(err) << ")" << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main()
{
    // 打印CUDA设备信息
    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, 0), "获取设备属性失败");
    std::cout << "使用设备: " << prop.name << std::endl;
    std::cout << "CUDA计算能力: " << prop.major << "." << prop.minor << std::endl;

    // 数组元素数量
    const int numElements = 50000;
    std::cout << "数组相加示例，每个数组包含 " << numElements << " 个元素" << std::endl;

    // 计算需要的线程块和每块线程数
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA配置: " << blocksPerGrid << " 个线程块 x " 
              << threadsPerBlock << " 个线程/块" << std::endl;

    // 分配主机内存（使用固定内存提高性能）
    float *h_A, *h_B, *h_C;
    checkCudaError(hipHostMalloc(&h_A, numElements * sizeof(float), hipHostMallocDefault), "分配固定主机内存A失败");
    checkCudaError(hipHostMalloc(&h_B, numElements * sizeof(float), hipHostMallocDefault), "分配固定主机内存B失败");
    checkCudaError(hipHostMalloc(&h_C, numElements * sizeof(float), hipHostMallocDefault), "分配固定主机内存C失败");

    // 初始化主机数组
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc(&d_A, numElements * sizeof(float)), "分配设备内存A失败");
    checkCudaError(hipMalloc(&d_B, numElements * sizeof(float)), "分配设备内存B失败");
    checkCudaError(hipMalloc(&d_C, numElements * sizeof(float)), "分配设备内存C失败");

    // 将数据从主机复制到设备
    std::cout << "将数据从主机内存复制到设备内存..." << std::endl;
    checkCudaError(hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice), 
                  "复制数据A到设备失败");
    checkCudaError(hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice), 
                  "复制数据B到设备失败");

    // 启动CUDA核函数
    std::cout << "启动CUDA核函数..." << std::endl;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    checkCudaError(hipGetLastError(), "内核启动失败");

    // 等待设备完成计算
    std::cout << "等待设备完成计算..." << std::endl;
    checkCudaError(hipDeviceSynchronize(), "设备同步失败");

    // 将结果从设备复制回主机
    std::cout << "将结果从设备内存复制回主机内存..." << std::endl;
    checkCudaError(hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost), 
                  "复制结果到主机失败");

    // 验证结果
    std::cout << "验证前5个结果..." << std::endl;
    for (int i = 0; i < 5; ++i)
    {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // 释放设备内存
    checkCudaError(hipFree(d_A), "释放设备内存A失败");
    checkCudaError(hipFree(d_B), "释放设备内存B失败");
    checkCudaError(hipFree(d_C), "释放设备内存C失败");

    // 释放主机内存
    checkCudaError(hipHostFree(h_A), "释放固定主机内存A失败");
    checkCudaError(hipHostFree(h_B), "释放固定主机内存B失败");
    checkCudaError(hipHostFree(h_C), "释放固定主机内存C失败");

    std::cout << "完成!" << std::endl;
    return 0;
}